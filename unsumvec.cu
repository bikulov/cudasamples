#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <math.h>
#include <iostream>

static const size_t N = 102400;

__global__ void kernel(const float* A, const float* B, float* C, int N)
{
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid < N) {
        C[tid] = A[tid] + B[tid];
    }
}

int main() {
    float *h_A, *h_B, *h_C;

    hipHostAlloc(&h_A, sizeof(float) * N, hipHostMallocDefault);
    hipHostAlloc(&h_B, sizeof(float) * N, hipHostMallocDefault);
    hipHostAlloc(&h_C, sizeof(float) * N, hipHostMallocDefault);
    
    for (int i = 0; i < N; i++) {
        h_A[i] = i;
        h_B[i] = 0.5f * i - 2;
    }

    float *d_A, *d_B, *d_C;
    hipHostGetDevicePointer(&d_A, h_A, 0);
    hipHostGetDevicePointer(&d_B, h_B, 0);
    hipHostGetDevicePointer(&d_C, h_C, 0);

    kernel<<<ceil(double(N) / 512), 512>>>(d_A, d_B, d_C, N);
    hipDeviceSynchronize();
  
    double err = 0;
    for (int i = 0; i < N; i++) {
        err += (h_A[i] + h_B[i]) - h_C[i];
    }
    std::cout << "Cum error: " << sqrt(err) << std::endl;
    
    hipHostFree(h_A);
    hipHostFree(h_B);
    hipHostFree(h_C);

    return 0;
}
